#include "hip/hip_runtime.h"
#include "header.hpp"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

InitializerBase::InitializerBase() {}

__global__ void setup_kernel(hiprandState *state, unsigned long seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, idx, idx, &state[idx]);
}

__global__ void check_rng(hiprandState* state, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float r1 = hiprand_uniform(&state[idx]);
        float r2 = hiprand_normal(&state[idx]);
        printf("Thread %d → uniform: %f, normal: %f\n", idx, r1, r2);
    }
}

XavierNormal::XavierNormal(): d_weights(nullptr) {}
XavierNormal::~XavierNormal() {
    if (d_weights) hipFree(d_weights);
}

__global__ void xavier_normal_initialize(hiprandState* state, float* d_weights, float multiplier, int total_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total_size) {
        d_weights[idx] = hiprand_normal(&state[idx]) * multiplier;
    }
}

float* XavierNormal::initialize(int shape_0, int shape_1) {
    int total_size = shape_0 * shape_1;
    float multiplier = sqrtf(2.0f / (shape_0 + shape_1));
    
    hipMalloc(&d_weights, total_size * sizeof(float));

    // set random
    hiprandState *d_state;
    hipMalloc(&d_state, total_size * sizeof(hiprandState));
    hipMemset(d_weights, 0, total_size * sizeof(float));
    hipMemset(d_state, 0, total_size * sizeof(hiprandState));

    auto now = chrono::high_resolution_clock::now();
    auto seed = chrono::duration_cast<chrono::microseconds>(now.time_since_epoch()).count();

    int threads_per_block = 256;
    int num_blocks = (total_size + threads_per_block - 1) / threads_per_block;
    setup_kernel<<<num_blocks, threads_per_block>>>(d_state, seed);
    hipDeviceSynchronize();
    xavier_normal_initialize<<<num_blocks, threads_per_block>>>(d_state, d_weights, multiplier, total_size);
    hipDeviceSynchronize();

    hipFree(d_state);

    return d_weights;
}

// ======================================================================================================================

XavierUniform::XavierUniform(): d_weights(nullptr) {}
XavierUniform::~XavierUniform() {
    if (d_weights) hipFree(d_weights);
}

__global__ void xavier_uniform_initialize(hiprandState* state, float* d_weights, float multiplier, int total_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total_size) {
        d_weights[idx] = 2*(hiprand_uniform(&state[idx])-0.5f) * multiplier;
    }
}

float* XavierUniform::initialize(int shape_0, int shape_1) {
    int total_size = shape_0 * shape_1;
    float multiplier = sqrtf(6.0f/(shape_0+shape_1));
    
    hipMalloc(&d_weights, total_size * sizeof(float));

    // set random
    hiprandState *d_state;
    hipMalloc(&d_state, total_size * sizeof(hiprandState));
    hipMemset(d_weights, 0, total_size * sizeof(float));
    hipMemset(d_state, 0, total_size * sizeof(hiprandState));

    auto now = std::chrono::high_resolution_clock::now();
    auto seed = std::chrono::duration_cast<std::chrono::microseconds>(now.time_since_epoch()).count();

    int threads_per_block = 256;
    int num_blocks = (total_size + threads_per_block - 1) / threads_per_block;
    setup_kernel<<<num_blocks, threads_per_block>>>(d_state, seed);
    hipDeviceSynchronize();
    xavier_uniform_initialize<<<num_blocks, threads_per_block>>>(d_state, d_weights, multiplier, total_size);
    hipDeviceSynchronize();

    hipFree(d_state);

    return d_weights;
}

// ======================================================================================================================

HeUniform::HeUniform(): d_weights(nullptr) {}
HeUniform::~HeUniform() {
    if (d_weights) hipFree(d_weights);
}

__global__ void he_uniform_initialize(hiprandState* state, float* d_weights, float multiplier, int total_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total_size) {
        d_weights[idx] = 2*(hiprand_uniform(&state[idx])-0.5f) * multiplier;
    }
}

float* HeUniform::initialize(int shape_0, int shape_1) {
    int total_size = shape_0 * shape_1;
    float multiplier = sqrtf(6.0f/(shape_0));
    
    hipMalloc(&d_weights, total_size * sizeof(float));

    // set random
    hiprandState *d_state;
    hipMalloc(&d_state, total_size * sizeof(hiprandState));
    hipMemset(d_weights, 0, total_size * sizeof(float));
    hipMemset(d_state, 0, total_size * sizeof(hiprandState));

    auto now = std::chrono::high_resolution_clock::now();
    auto seed = std::chrono::duration_cast<std::chrono::microseconds>(now.time_since_epoch()).count();

    int threads_per_block = 256;
    int num_blocks = (total_size + threads_per_block - 1) / threads_per_block;
    setup_kernel<<<num_blocks, threads_per_block>>>(d_state, seed);
    hipDeviceSynchronize();
    he_uniform_initialize<<<num_blocks, threads_per_block>>>(d_state, d_weights, multiplier, total_size);
    hipDeviceSynchronize();

    hipFree(d_state);

    return d_weights;
}

// ======================================================================================================================

HeNormal::HeNormal(): d_weights(nullptr) {}
HeNormal::~HeNormal() {
    if (d_weights) hipFree(d_weights);
}

__global__ void he_normal_initialize(hiprandState* state, float* d_weights, float multiplier, int total_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total_size) {
        d_weights[idx] = hiprand_normal(&state[idx]) * multiplier;
    }
}

float* HeNormal::initialize(int shape_0, int shape_1) {
    int total_size = shape_0 * shape_1;
    float multiplier = sqrtf(2.0f/(shape_0));
    
    hipMalloc(&d_weights, total_size * sizeof(float));

    // set random
    hiprandState *d_state;
    hipMalloc(&d_state, total_size * sizeof(hiprandState));
    hipMemset(d_weights, 0, total_size * sizeof(float));
    hipMemset(d_state, 0, total_size * sizeof(hiprandState));

    auto now = std::chrono::high_resolution_clock::now();
    auto seed = std::chrono::duration_cast<std::chrono::microseconds>(now.time_since_epoch()).count();

    int threads_per_block = 256;
    int num_blocks = (total_size + threads_per_block - 1) / threads_per_block;
    setup_kernel<<<num_blocks, threads_per_block>>>(d_state, seed);
    hipDeviceSynchronize();
    he_normal_initialize<<<num_blocks, threads_per_block>>>(d_state, d_weights, multiplier, total_size);
    hipDeviceSynchronize();

    hipFree(d_state);

    return d_weights;
}