#include "hip/hip_runtime.h"
#include "header.hpp"

ActivationFuncBase::ActivationFuncBase() {}

Sigmoid::Sigmoid() : output(), d_input(nullptr), d_output(nullptr), current_size(0) {}

Sigmoid::~Sigmoid() {
    if (d_input) hipFree(d_input);
    if (d_output) hipFree(d_output);
    if (d_grad) hipFree(d_grad);
    if (d_backward_result) hipFree(d_backward_result);
}

__global__ void sigmoid_kernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float x = input[idx];
        output[idx] = 1.0f / (1.0f + expf(-x));
    }
}

vector<vector<float>> Sigmoid::forward(vector<vector<float>>& pre_activation_values) {
    int batch_size = pre_activation_values.size();
    int feature_size = pre_activation_values[0].size();
    int total_size = batch_size * feature_size;

    vector<float> flat_input(total_size);
    flat_input = flatten(pre_activation_values);

    // Allocate GPU memory if size changed
    if (total_size > current_size) {
        if (d_input) hipFree(d_input);
        if (d_output) hipFree(d_output);
        hipMalloc(&d_input, total_size * sizeof(float));
        hipMalloc(&d_output, total_size * sizeof(float));
        current_size = total_size;
    }

    // Copy to device
    hipMemcpy(d_input, flat_input.data(), total_size * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    int threads_per_block = 256;
    int num_blocks = (total_size + threads_per_block - 1) / threads_per_block;
    sigmoid_kernel<<<num_blocks, threads_per_block>>>(d_input, d_output, total_size);
    hipDeviceSynchronize();

    // Copy result back
    vector<float> flat_output(total_size);
    hipMemcpy(flat_output.data(), d_output, total_size * sizeof(float), hipMemcpyDeviceToHost);

    // Reshape to 2D
    vector<vector<float>> result(batch_size, vector<float>(feature_size));
    result = unflatten(flat_output, batch_size, feature_size);

    // Store 1D output for backward()
    output = move(flat_output);

    return result;
}

__global__ void sigmoid_backward_kernel(const float* grad,  const float* output, float* result, int batch_size, int output_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_size = batch_size * output_size;

    if (idx < total_size) {
        int j = idx % output_size;
        result[idx] = grad[idx] * output[j] * (1.0f - output[j]);
    }
}

vector<vector<float>> Sigmoid::backward(vector<vector<float>>& gradient) {
    int batch_size = gradient.size();
    int output_size = gradient[0].size();
    int total_size = batch_size * output_size;

    // Resize GPU buffers only if needed
    if (batch_size != last_batch_size || output_size != current_size) {
        if (d_grad) hipFree(d_grad);
        if (d_backward_result) hipFree(d_backward_result);

        hipMalloc(&d_grad, total_size * sizeof(float));
        hipMalloc(&d_backward_result, total_size * sizeof(float));
        last_batch_size = batch_size;
        current_size = output_size; 
    }

    // Flatten gradient to 1D (2D gay)
    vector<float> flat_grad(total_size);
    flat_grad = flatten(gradient);

    // Copy inputs to device
    hipMemcpy(d_grad, flat_grad.data(), total_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_output, output.data(), output_size * sizeof(float), hipMemcpyHostToDevice);  // already allocated in forward

    // Launch kernel
    int threads = 256;
    int blocks = (total_size + threads - 1) / threads;
    sigmoid_backward_kernel<<<blocks, threads>>>(d_grad, d_output, d_backward_result, batch_size, output_size);
    hipDeviceSynchronize();

    // Copy result back
    vector<float> flat_result(total_size);
    hipMemcpy(flat_result.data(), d_backward_result, total_size * sizeof(float), hipMemcpyDeviceToHost);

    // Reshape to 2D
    vector<vector<float>> result(batch_size, vector<float>(output_size));
    result = unflatten(flat_result, batch_size, output_size);

    return result;
}